#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"

using namespace optix;

rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
  const float t = max(dot(ray.direction, up), 0.0f);
  const float3 result = lerp(background_light, background_dark, t);

  prd_radiance.radiance = result;
  prd_radiance.done = true;
}
