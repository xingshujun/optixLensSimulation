#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "intersection_refinement.h"
#define Pi      3.14159265358979f
#define eps     1.0e-5f

using namespace optix;

//�����б�
rtDeclareVariable(float,  radius, , );
rtDeclareVariable(float,  partialAngle, , );
rtDeclareVariable(float,  zMin, , );
rtDeclareVariable(float,  zMax, , );
rtDeclareVariable(float,  phiMax, , );
rtDeclareVariable(float,  thetaMax, , );
rtDeclareVariable(float,  thetaMin, , );


//�����б�
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );  
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );

//ֻ���б�
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

static __device__  float clampT(const float f, const float a, const float b)
{
	return fmaxf(a, fminf(f, b));
}
//�����������ʽ�ı��
static __device__ bool Quadratic(float a, float b, float c, float &t0, float &t1)
{
	// Find quadratic discriminant
	float discrim = b * b - 4 * a * c;
	if (discrim < 0) 
		return false;

	float rootDiscrim = sqrtf(discrim);

	// Compute quadratic _t_ values
	float q;
	if (b < 0)
		q = -.5 * (b - rootDiscrim);
	else
		q = -.5 * (b + rootDiscrim);
	t0 = q / a;
	t1 = c / q;
	float ttemp = 0;

	if (t0 > t1) 
	{
		ttemp = t0;
		t0 = t1;
		t1 = ttemp;
	}

	return true;
}

static __device__  float UpperBound(float v)
{
	float err = eps;
	return v + err;
}

static __device__  float LowerBound(float v)
{
	float err = eps;
	return v - err;
}


//�ο����ӣ�https://github.com/mmp/pbrt-v3/blob/6663b4cd4cb242ef8b923d99a6cb0b27b2fb0f37/src/shapes/sphere.cpp

RT_PROGRAM void partialsphere_intersect(int primIdx)
{
	float phi;   
	float3 dir = ray.direction;
	float3 ori = ray.origin;

	// ��ʼ�����ߵ�����ֵ
	float tmin = ray.tmin;
	float tmax = ray.tmax;

	// ������Ӧ�Ķ��η��̲���
	float a = dot(dir, dir);
	float b = 2.0 * dot(dir, ori);
	float c = dot(ori, ori) - radius*radius;

	// ����η�����ⷽ������Tֵ
	float t0, t1;
	if (!Quadratic(a, b, c, t0, t1)) 
		return ;

	// ���t0,t1�Ƿ����������ײ��
	if (UpperBound(t0) > tmax || LowerBound(t1) <= 0)
		return ;

	float tShapeHit = t0;
	if (UpperBound(tShapeHit) <= 0)
	{
		tShapeHit = t1;
		if (UpperBound(tShapeHit) > tmin)
			return ;
	}

	// ������������ײ����phiֵ�Ĵ�С
	float3 pHit = ori + tShapeHit * dir;

	// ������������ߵ���ײ��
	pHit = radius / length(pHit)*pHit;

	if (pHit.x == 0 && pHit.y == 0)
		pHit.x = 1e-5f * radius;

	phi = atan2(pHit.y, pHit.x);

	if (phi < 0) 
		phi += 2 * Pi;

	// �Բ����������ײ��ײ������
	if ((zMin > -radius && pHit.z < zMin) || (zMax < radius && pHit.z > zMax) ||phi > phiMax) 
	{
		if (tShapeHit == t1) 
			return ;
		if (UpperBound(t1) > tmax)
			return ;
		tShapeHit = t1;
		// ����������ײ���
		pHit = ori + tShapeHit * dir;

		// ����������ײ��
		pHit = radius / length(pHit)*pHit;
		if (pHit.x == 0 && pHit.y == 0) pHit.x = 1e-5f * radius;
		phi = atan2(pHit.y, pHit.x);
		if (phi < 0) phi += 2 * Pi;
		if ((zMin > -radius && pHit.z < zMin) ||
			(zMax < radius && pHit.z > zMax) || phi > phiMax)
			return ;
	}

	// ʹ�ò������ķ�ʽ���������
	float u = phi / phiMax;
	float theta = acosf(clampT(pHit.z / radius, -1, 1));
	float v = (theta - thetaMin) / (thetaMax - thetaMin);

	if (rtPotentialIntersection(tShapeHit))
	{
		geometric_normal = shading_normal = pHit;
		if (dot(dir, geometric_normal) > 0.0f) 
		{
			back_hit_point  = offset(pHit,  geometric_normal);
			front_hit_point = offset(pHit, -geometric_normal);
		}
		else 
		{
			back_hit_point  = offset(pHit, -geometric_normal);
			front_hit_point = offset(pHit,  geometric_normal);
		}

		rtReportIntersection(0);
	}



}


RT_PROGRAM void partialsphere_bounds(int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;
	float anchor = 3.0f;
	aabb->m_min = make_float3(anchor, anchor, anchor);
	aabb->m_max = make_float3(anchor, anchor, anchor);
}
