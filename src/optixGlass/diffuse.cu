#include "hip/hip_runtime.h"
///������ʾ
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable( float2, Kd_map_scale, , );

RT_PROGRAM void closest_hit_radiance()
{

    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    const float z1 = rnd( prd_radiance.seed );
    const float z2 = rnd( prd_radiance.seed );
    
    float3 w_in;
    optix::cosine_sample_hemisphere( z1, z2, w_in );
    const optix::Onb onb( ffnormal );
    onb.inverse_transform( w_in );
    const float3 fhp = rtTransformPoint( RT_OBJECT_TO_WORLD, front_hit_point );

    prd_radiance.origin = front_hit_point;
    prd_radiance.direction = w_in;
    
    const float3 Kd = make_float3( tex2D( Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y ) );
    prd_radiance.reflectance *= Kd;

}

